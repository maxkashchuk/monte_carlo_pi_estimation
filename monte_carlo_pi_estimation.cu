#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>
#include <math.h>

__global__ void generateXORWOW(float *d_data, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandStateXORWOW_t state;
    hiprand_init(1234, tid, 0, &state);

    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        float randomValue = hiprand_uniform(&state);

        d_data[i] = 2.0f * randomValue - 1.0f;
    }
}

__global__ void generateMRG32k3a(float *d_data, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandStateMRG32k3a_t state;
    hiprand_init(2134, tid, 0, &state);

    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        float randomValue = hiprand_uniform(&state);

        d_data[i] = 2.0f * randomValue - 1.0f;
    }
}

__global__ void generatePhilox_4x32_10(float *d_data, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandStatePhilox4_32_10_t state;
    hiprand_init(3124, tid, 0, &state);

    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        float randomValue = hiprand_uniform(&state);

        d_data[i] = 2.0f * randomValue - 1.0f;
    }
}

__global__ void generateSobol(float *d_data, hiprandDirectionVectors32_t *directionVectors, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandStateSobol32_t state;

    hiprand_init(*directionVectors, tid, &state);

    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        float randomValue = hiprand_uniform(&state);

        d_data[i] = 2.0f * randomValue - 1.0f;
    }
}

void calculate_pi(float *d_data_1, float *d_data_2, int n)
{
    float counter = 0.0f;

    float pi = 0.0f;

    for (int i = 0; i < n; i++)
    {
        // std::cout << "Counter first pair: " << d_data_1[i] << "; " << d_data_2[i] << std::endl;
        if(((d_data_1[i] * d_data_1[i]) + (d_data_2[i] * d_data_2[i])) <= 1.0f)
        {
            counter++;
        }
    }

    std::cout << "Counter value: " << counter << std::endl;

    pi = 4.0f * (counter / (float)n);

    std::cout << "PI value: " << pi << std::endl;
}

int main(void)
{
    const int n = 1000000;
    const int blockSize = 256;
    const int numBlocks = (n + blockSize - 1) / blockSize;

    float *h_data_1 = new float[n];
    float *d_data_1;

    float *h_data_2 = new float[n];
    float *d_data_2;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // XORWOW

    hipMalloc((void**)&d_data_1, n * sizeof(float));
    hipMalloc((void**)&d_data_2, n * sizeof(float));

    hipEventRecord(start);
    generateXORWOW<<<numBlocks, blockSize>>>(d_data_1, n);

    generateXORWOW<<<numBlocks, blockSize>>>(d_data_2, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Execution time XORWOW: " << milliseconds << " ms" << std::endl;

    hipMemcpy(h_data_1, d_data_1, n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_data_2, d_data_2, n * sizeof(float), hipMemcpyDeviceToHost);

    calculate_pi(h_data_1, h_data_2, n);

    delete[] h_data_1;
    delete[] h_data_2;
    hipFree(d_data_1);
    hipFree(d_data_2);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // MRG32k3a

    hipEventCreate(&start);
    hipEventCreate(&stop);

    h_data_1 = new float[n];
    h_data_2 = new float[n];

    hipMalloc((void**)&d_data_1, n * sizeof(float));
    hipMalloc((void**)&d_data_2, n * sizeof(float));

    hipEventRecord(start);
    generateMRG32k3a<<<numBlocks, blockSize>>>(d_data_1, n);

    generateMRG32k3a<<<numBlocks, blockSize>>>(d_data_2, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Execution time MRG32k3a: " << milliseconds << " ms" << std::endl;

    hipMemcpy(h_data_1, d_data_1, n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_data_2, d_data_2, n * sizeof(float), hipMemcpyDeviceToHost);

    calculate_pi(h_data_1, h_data_2, n);

    delete[] h_data_1;
    delete[] h_data_2;
    hipFree(d_data_1);
    hipFree(d_data_2);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Philox_4x32_10

    hipEventCreate(&start);
    hipEventCreate(&stop);

    h_data_1 = new float[n];
    h_data_2 = new float[n];

    hipMalloc((void**)&d_data_1, n * sizeof(float));
    hipMalloc((void**)&d_data_2, n * sizeof(float));

    hipEventRecord(start);
    generatePhilox_4x32_10<<<numBlocks, blockSize>>>(d_data_1, n);

    generatePhilox_4x32_10<<<numBlocks, blockSize>>>(d_data_2, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Execution time Philox_4x32_10: " << milliseconds << " ms" << std::endl;

    hipMemcpy(h_data_1, d_data_1, n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_data_2, d_data_2, n * sizeof(float), hipMemcpyDeviceToHost);

    calculate_pi(h_data_1, h_data_2, n);

    delete[] h_data_1;
    delete[] h_data_2;
    hipFree(d_data_1);
    hipFree(d_data_2);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Sobol

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hiprandDirectionVectors32_t *directionVectors;

    const size_t numDirectionVectors = 32;

    hipHostMalloc((void**)&directionVectors, numDirectionVectors * 20000, hipHostMallocDefault);

    hiprandGetDirectionVectors32(&directionVectors, HIPRAND_DIRECTION_VECTORS_32_JOEKUO6);

    h_data_1 = new float[n];
    h_data_2 = new float[n];

    hipMalloc((void**)&d_data_1, n * sizeof(float));
    hipMalloc((void**)&d_data_2, n * sizeof(float));

    hipEventRecord(start);
    generateSobol<<<numBlocks, blockSize>>>(d_data_1, directionVectors, n);

    generateSobol<<<numBlocks, blockSize>>>(d_data_2, directionVectors, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Execution time Sobol: " << milliseconds << " ms" << std::endl;

    hipMemcpy(h_data_1, d_data_1, n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_data_2, d_data_2, n * sizeof(float), hipMemcpyDeviceToHost);

    calculate_pi(h_data_1, h_data_2, n);

    delete[] h_data_1;
    delete[] h_data_2;
    hipFree(d_data_1);
    hipFree(d_data_2);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipHostFree(directionVectors);

    return 0;
}
